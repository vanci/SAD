#include "hip/hip_runtime.h"
#include <hip/device_functions.h>
#include "hip/hip_runtime.h"
#include ""

#include <memory>
#include <cassert>

#include "SAD.h"

#define KERNEL_LENGTH	10000000
#define CUDA_CHECK(x,y)  if((x) != hipSuccess){ puts(y); assert(0); }
#define MIN(a,b)	((a)<(b)?(a):(b))

// MemoryManager allocates memory for Jacobian computation
void MemoryManagerForward::Allocate(int M, int N)
{
	this->M = M;
	this->N = N;

	// Expected number of variables
	nvar_S = M-1;
	nvar_K = nvar_S + 3*(M-1);
	nvar_T0  = nvar_K + (M-1) + 4;
	nvar_T = nvar_T0 + (M-1)*M*M;

	// Expected number of arcs in the DAG
	nnzpd_S = 0;
	nnzpd_K = 5 + (M-3)*7 + 5;
	nnzpd_T0 = nnzpd_K;
	nnzpd_T = nnzpd_T0 + (M-1)*M*M*6;

	assert(ADS::nvar == nvar_T);	// actual number of variables
	assert(ADS::nnz_pd == nnzpd_T);	// actual number of arcs in the DAG

	hipError_t err = hipSuccess;

	err = hipMalloc(&D1, sizeof(*D1)*(1+M*(M-1)));
	CUDA_CHECK(err, "D1 allocation failed.");
	hipMemset(D1, 0, sizeof(*D1)*(1+M*(M-1)));
	d1 = D1 + 1;

	err = hipMalloc(&D2, sizeof(*D1)*(1+M*(M-1)));
	CUDA_CHECK(err, "D2 allocation failed.");
	hipMemset(D2, 0, sizeof(*D1)*(1+M*(M-1)));
	d2 = D2 + 1;

	err = hipMalloc(&dK, sizeof(*dK)*(nvar_K-nvar_S)*(M-1));
	CUDA_CHECK(err, "dK allocation failed.");
	h_dK = new float[(nvar_K-nvar_S)];
	memset(h_dK, 0, sizeof(*h_dK)*(nvar_K-nvar_S));

	err = hipMalloc(&pd, sizeof(*pd)*ADS::nnz_pd);
	CUDA_CHECK(err,"pd allocation failed.");
	err = hipMemcpy(pd, ADS::pd, sizeof(*pd)*ADS::nnz_pd, hipMemcpyHostToDevice);
	CUDA_CHECK(err,"pd memcpy failed.");
		
}

void MemoryManagerForward::Clear()
{
	hipFree(D1);
	hipFree(D2);
	hipFree(dK);
	hipFree(pd);

	delete [] h_dK;	
}

__global__ void propagateKernel(float *p1, float *p2, float *dK, float *pd, int M, int N, int b)
{
	int j = threadIdx.x;
	int k = 6*j;

	float *d1 = p1 + blockIdx.x * M;
	float *d2 = p2 + blockIdx.x * M;
	float *t, *result;

	float dK_row[3];
	int m = 3*j + blockIdx.x * 3 * (M-1);
	dK_row[0] = dK[m];
	dK_row[1] = dK[m+1];
	dK_row[2] = dK[m+2];

	for(int n = 0; n < N; ++n)
	{
		d2[j] = pd[k] * dK_row[0] + pd[k+1] * dK_row[1] + pd[k+2] * dK_row[2]
			+ pd[k+3] * d1[j-1] + pd[k+4] * d1[j] + pd[k+5] * d1[j+1];
		k += b;

		t = d2, d2 = d1, d1 = t;
		__syncthreads();
	}
	p1[blockIdx.x * M + j] = d1[j];
}

void ADS::cudaGetJacobianForward(float *J, int m, MemoryManagerForward &mmf)
{
	int M = mmf.M;
	int N = mmf.N;
	int NOI = M - 1;

	hipError_t err = hipSuccess;
	float *dK = mmf.dK,
		*h_dK = mmf.h_dK;

	float *d1 = mmf.d1;
	float *d2 = mmf.d2;

	int *rid = ADS::cooRow;
	int *cid = ADS::cooCol;
	float *pd = mmf.pd, *h_pd = ADS::pd;

	for(int xid = 0; xid < NOI; ++xid)
	{
		for(int ipd = 0; ipd < mmf.nnzpd_K; ++ipd)
		{
			if( cid[ipd] == xid )
				h_dK[ rid[ipd] - mmf.nvar_S ] += h_pd[ipd];
		}

		err = hipMemcpy(dK + xid*3*(M-1), h_dK, sizeof(*h_dK)*3*(M-1), hipMemcpyHostToDevice);
		CUDA_CHECK(err,"dK memcpy failed.");
		hipDeviceSynchronize();
		memset(h_dK, 0, sizeof(*h_dK)*3*(M-1));
	}

	int C = (M-1)*6;
	int StepSize = KERNEL_LENGTH / C;
	for(int n = 0; n < N; n+=StepSize)
	{
		int n_ceil = MIN(N-n, StepSize);
		propagateKernel<<<m,mmf.nvar_S>>>(d1, d2, dK, pd+mmf.nnzpd_K + C*n, M, n_ceil, C);
		err = hipDeviceSynchronize();
		CUDA_CHECK(err,"Kernel error");
	}

	for(int j = 0; j < NOI; ++j)
	{
		err = hipMemcpy(J+NOI*j, d1+M*j, sizeof(*J)*NOI, hipMemcpyDeviceToHost);
		CUDA_CHECK(err,"J memcpy failed.");
	}
}